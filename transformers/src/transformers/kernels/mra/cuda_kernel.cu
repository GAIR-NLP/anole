#include "hip/hip_runtime.h"
#include "cuda_kernel.h"

//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void index_max_cuda_kernel(
  float *index_vals,       // [batch_size, 32, num_block]
  int   *indices,        // [batch_size, num_block]
  float *max_vals,        // [batch_size, A_num_block * 32]
  float *max_vals_scatter,   // [batch_size, 32, num_block]
  long batch_size,
  long A_num_block,
  long B_num_block,
  long num_block
) {

  long batch_idx = blockIdx.x;

  long thread_idx = threadIdx.x;
  long num_thread = blockDim.x;

  extern __shared__ float buffer[];
  int *max_buffer = (int*)buffer;

  for (int i = 0; i < A_num_block * 32; i = i + num_thread) {
    int idx = i + thread_idx;
    if (idx < A_num_block * 32) {
      max_buffer[idx] = -1e8;
    }
  }
  __syncthreads();

  int *indices_pt = &indices[batch_idx * num_block];
  float *index_vals_pt = &index_vals[batch_idx * num_block * 32];

  for (int idx_start = 0; idx_start < 32 * num_block; idx_start = idx_start + num_thread) {
    int idx = idx_start + thread_idx;
    int A_block_idx = indices_pt[idx % num_block] / B_num_block;
    atomicMax(&max_buffer[A_block_idx * 32 + idx / num_block], (int)(index_vals_pt[idx] * 1000));
  }
  __syncthreads();
  
  float *max_vals_pt = &max_vals[batch_idx * A_num_block * 32];
  for (int i = 0; i < A_num_block * 32; i = i + num_thread) {
    int idx = i + thread_idx;
    if (idx < A_num_block * 32) {
      max_vals_pt[idx] = (float)max_buffer[idx] / 1000.;
    }
  }
  
  float *max_vals_scatter_pt = &max_vals_scatter[batch_idx * num_block * 32];
  for (int idx_start = 0; idx_start < 32 * num_block; idx_start = idx_start + num_thread) {
    int idx = idx_start + thread_idx;
    int A_block_idx = indices_pt[idx % num_block] / B_num_block;
    max_vals_scatter_pt[idx] = (float)max_buffer[A_block_idx * 32 + idx / num_block] / 1000.;
  }

}

__global__ void mm_to_sparse_cuda_kernel(
  float *dense_A,   // [batch_size, A_num_block, dim, 32]
  float *dense_B,   // [batch_size, B_num_block, dim, 32]
  int   *indices,   // [batch_size, num_block]
  float *sparse_C,  // [batch_size, num_block, 32, 32]
  long batch_size,
  long A_num_block,
  long B_num_block,
  long dim,
  long num_block
) {

  long batch_idx = blockIdx.y;
  long block_idx = blockIdx.x * blockDim.y + threadIdx.y;

  long thread_idx = threadIdx.x;

  __shared__ float buffer[4096];
  float *A_buffer = &buffer[threadIdx.y * 1024]; // [2, 8, 32]
  float *B_buffer = &buffer[threadIdx.y * 1024 + 512]; // [2, 8, 32]

  long batch_idx__block_idx = batch_idx * num_block + block_idx;

  long AB_block_idx = indices[batch_idx__block_idx];
  float *dense_A_pt = &dense_A[(batch_idx * A_num_block + AB_block_idx / B_num_block) * dim * 32];
  float *dense_B_pt = &dense_B[(batch_idx * B_num_block + AB_block_idx % B_num_block) * dim * 32];

  int reg_1_idx = thread_idx / 8;    // [0000000011111111222222223333333344444444555555556666666677777777]
  int reg_2_idx = thread_idx % 8;    // [0123456701234567012345670123456701234567012345670123456701234567]

  float reg_1[8];
  float reg_2[8];

  float reg_array[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  #pragma unroll
  for (int i = 0; i < 4; i++) {
    A_buffer[i * 64 + thread_idx] = dense_A_pt[i * 64 + thread_idx];
    B_buffer[i * 64 + thread_idx] = dense_B_pt[i * 64 + thread_idx];
  }

  __syncthreads();

  #pragma unroll
  for (int i = 0; i < 4; i++) {
    reg_1[i] = A_buffer[reg_1_idx * 4 + i];
    reg_2[i] = B_buffer[reg_2_idx * 4 + i];
  }

  for (int dim_stride = 1; dim_stride < (dim / 8); dim_stride++) {

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      A_buffer[(dim_stride % 2) * 256 + i * 64 + thread_idx] = dense_A_pt[dim_stride * 256 + i * 64 + thread_idx];
      B_buffer[(dim_stride % 2) * 256 + i * 64 + thread_idx] = dense_B_pt[dim_stride * 256 + i * 64 + thread_idx];
    }

    #pragma unroll
    for (int mini_dim_idx = 1; mini_dim_idx < 8; mini_dim_idx++) {
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        reg_1[(mini_dim_idx % 2) * 4 + i] = A_buffer[((dim_stride - 1) % 2) * 256 + mini_dim_idx * 32 + reg_1_idx * 4 + i];
        reg_2[(mini_dim_idx % 2) * 4 + i] = B_buffer[((dim_stride - 1) % 2) * 256 + mini_dim_idx * 32 + reg_2_idx * 4 + i];
      }
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          reg_array[i * 4 + j] += reg_1[((mini_dim_idx - 1) % 2) * 4 + i] * reg_2[((mini_dim_idx - 1) % 2) * 4 + j];
        }
      }
    }

    __syncthreads();

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      reg_1[i] = A_buffer[(dim_stride % 2) * 256 + reg_1_idx * 4 + i];
      reg_2[i] = B_buffer[(dim_stride % 2) * 256 + reg_2_idx * 4 + i];
    }

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      #pragma unroll
      for (int j = 0; j < 4; j++) {
        reg_array[i * 4 + j] += reg_1[4 + i] * reg_2[4 + j];
      }
    }

  }

  #pragma unroll
  for (int mini_dim_idx = 1; mini_dim_idx < 8; mini_dim_idx++) {
    #pragma unroll
    for (int i = 0; i < 4; i++) {
      reg_1[(mini_dim_idx % 2) * 4 + i] = A_buffer[256 + mini_dim_idx * 32 + reg_1_idx * 4 + i];
      reg_2[(mini_dim_idx % 2) * 4 + i] = B_buffer[256 + mini_dim_idx * 32 + reg_2_idx * 4 + i];
    }
    #pragma unroll
    for (int i = 0; i < 4; i++) {
      #pragma unroll
      for (int j = 0; j < 4; j++) {
        reg_array[i * 4 + j] += reg_1[((mini_dim_idx - 1) % 2) * 4 + i] * reg_2[((mini_dim_idx - 1) % 2) * 4 + j];
      }
    }
  }
  #pragma unroll
  for (int i = 0; i < 4; i++) {
    #pragma unroll
    for (int j = 0; j < 4; j++) {
      reg_array[i * 4 + j] += reg_1[4 + i] * reg_2[4 + j];
    }
  }
  __syncthreads();

  float *C_buffer = &buffer[threadIdx.y * 1024]; // [32, 32]

  #pragma unroll
  for (int i = 0; i < 4; i++) {
    #pragma unroll
    for (int j = 0; j < 4; j++) {
      C_buffer[(reg_2_idx * 4 + j) * 32 + reg_1_idx * 4 + i] = reg_array[i * 4 + j];
    }
  }
  __syncthreads();

  float *sparse_C_pt = &sparse_C[batch_idx__block_idx * 1024];

  #pragma unroll
  for (int i = 0; i < 16; i++) {
    sparse_C_pt[i * 64 + thread_idx] = C_buffer[i * 64 + thread_idx];
  }

}

__global__ void sparse_dense_mm_cuda_kernel(
  float *sparse_A,  // [batch_size, num_block, 32, 32]
  int   *indices,   // [batch_size, num_block]
  float *dense_B,   // [batch_size, B_num_block, dim, 32]
  float *dense_C,   // [batch_size, A_num_block, dim, 32]
  long batch_size,
  long A_num_block,
  long B_num_block,
  long dim,
  long num_block
) {

  long batch_idx = blockIdx.y;
  long block_idx = blockIdx.x * blockDim.y + threadIdx.y;

  long thread_idx = threadIdx.x;

  __shared__ float buffer[6144];
  float *A_buffer = &buffer[threadIdx.y * 3072]; // [32, 32]
  float *B_buffer = &buffer[threadIdx.y * 3072 + 1024]; // [32, 64]

  long batch_idx__block_idx = batch_idx * num_block + block_idx;

  float *sparse_A_pt = &sparse_A[batch_idx__block_idx * 1024];
  #pragma unroll
  for (int i = 0; i < 8; i++) {
    A_buffer[i * 128 + thread_idx] = sparse_A_pt[i * 128 + thread_idx];
  }

  long AB_block_idx = indices[batch_idx__block_idx];
  float *dense_B_pt = &dense_B[(batch_idx * B_num_block + AB_block_idx % B_num_block) * 32 * dim];
  float *dense_C_pt = &dense_C[(batch_idx * A_num_block + AB_block_idx / B_num_block) * 32 * dim];

  // [0000000011111111222222223333333344444444555555556666666677777777]
  // [0123456701234567012345670123456701234567012345670123456701234567]
  int reg_1_idx = thread_idx / 8;
  int reg_2_idx = thread_idx % 8;

  float reg_1[8];
  float reg_2[8];

  float reg_array[16];

  for (int dim_stride = 0; dim_stride < dim; dim_stride = dim_stride + 64) {

    #pragma unroll
    for (int i = 0; i < 16; i++) {
      B_buffer[i * 128 + thread_idx] = dense_B_pt[dim_stride * 32 + i * 128 + thread_idx];
    }

    #pragma unroll
    for (int i = 0; i < 16; i++) {
      reg_array[i] = 0;
    }

    __syncthreads();

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      reg_1[i] = B_buffer[(reg_1_idx * 4 + i) * 32];
      reg_2[i] = A_buffer[reg_2_idx * 4 + i];
    }

    #pragma unroll
    for (int mini_dim_idx = 1; mini_dim_idx < 32; mini_dim_idx++) {
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        reg_1[(mini_dim_idx % 2) * 4 + i] = B_buffer[(reg_1_idx * 4 + i) * 32 + mini_dim_idx];
        reg_2[(mini_dim_idx % 2) * 4 + i] = A_buffer[mini_dim_idx * 32 + reg_2_idx * 4 + i];
      }
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          reg_array[i * 4 + j] += reg_1[((mini_dim_idx - 1) % 2) * 4 + i] * reg_2[((mini_dim_idx - 1) % 2) * 4 + j];
        }
      }
    }

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      #pragma unroll
      for (int j = 0; j < 4; j++) {
        reg_array[i * 4 + j] += reg_1[4 + i] * reg_2[4 + j];
      }
    }

    __syncthreads();

    float *C_buffer = &buffer[threadIdx.y * 3072 + 1024]; // [64, 32]

    #pragma unroll
    for (int i = 0; i < 4; i++) {
      #pragma unroll
      for (int j = 0; j < 4; j++) {
        C_buffer[(reg_1_idx * 4 + i) * 32 + reg_2_idx * 4 + j] = reg_array[i * 4 + j];
      }
    }
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < 16; i++) {
      atomicAdd(&dense_C_pt[dim_stride * 32 + i * 128 + thread_idx], C_buffer[i * 128 + thread_idx]);
    }
    __syncthreads();

  }

}


__global__ void reduce_sum_cuda_kernel(
  float *sparse_A,  // [batch_size, num_block, 32, 32]
  int   *indices,   // [batch_size, num_block]
  float *dense_C,   // [batch_size, A_num_block, 32]
  long batch_size,
  long A_num_block,
  long B_num_block,
  long num_block
) {

  long batch_idx = blockIdx.y;
  long block_idx = blockIdx.x * blockDim.y + threadIdx.y;

  long thread_idx = threadIdx.x;

  long batch_idx__block_idx = batch_idx * num_block + block_idx;

  long AB_block_idx = indices[batch_idx__block_idx];
  float *sparse_A_pt = &sparse_A[batch_idx__block_idx * 1024];

  float reg_array[16];
  float value = 0;

  #pragma unroll
  for (int i = 0; i < 8; i++) {
    reg_array[i] = sparse_A_pt[i * 32 + thread_idx];
  }
  #pragma unroll
  for (int stride = 8; stride < 32; stride = stride + 8) {
    #pragma unroll
    for (int i = 0; i < 8; i++) {
      reg_array[(stride + i) % 16] = sparse_A_pt[(stride + i) * 32 + thread_idx];
    }
    #pragma unroll
    for (int i = 0; i < 8; i++) {
      value = value + reg_array[(stride - 8 + i) % 16];
    }
  }
  #pragma unroll
  for (int i = 0; i < 8; i++) {
    value = value + reg_array[8 + i];
  }

  float *dense_C_pt = &dense_C[(batch_idx * A_num_block + AB_block_idx / B_num_block) * 32];

  atomicAdd(&dense_C_pt[thread_idx], value);

}

__global__ void scatter_cuda_kernel(
  float *dense_A,   // [batch_size, A_num_block, 32]
  int   *indices,   // [batch_size, num_block]
  float *sparse_C,  // [batch_size, num_block, 32, 32]
  long batch_size,
  long A_num_block,
  long B_num_block,
  long num_block
) {

  long batch_idx = blockIdx.y;
  long block_idx = blockIdx.x * blockDim.y + threadIdx.y;

  long thread_idx = threadIdx.x;

  long batch_idx__block_idx = batch_idx * num_block + block_idx;

  long AB_block_idx = indices[batch_idx__block_idx];
  float *dense_A_pt = &dense_A[(batch_idx * A_num_block + AB_block_idx / B_num_block) * 32];
  float *sparse_C_pt = &sparse_C[(batch_idx * num_block + block_idx) * 1024];

  float value = dense_A_pt[thread_idx];

  #pragma unroll
  for (int i = 0; i < 32; i++) {
    sparse_C_pt[i * 32 + thread_idx] = value;
  }

}
